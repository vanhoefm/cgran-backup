#include "hip/hip_runtime.h"
/* -*- cuda -*- */
/*
 * Copyright 2011 Free Software Foundation, Inc.
 * 
 * This file is part of GNU Radio
 * 
 * GNU Radio is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 3, or (at your option)
 * any later version.
 * 
 * GNU Radio is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with GNU Radio; see the file COPYING.  If not, write to
 * the Free Software Foundation, Inc., 51 Franklin Street,
 * Boston, MA 02110-1301, USA.
 */ 

#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <cutil_inline.h>
#include <grgpu_utils.h>


__global__ void sub_const_ff_cuda_kernel(float* d_idata, float *d_odata, float k, int n)
{
  int idx = threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<n)
    d_odata[idx]=d_idata[idx]+k;
}


void grgpu_sub_const_ff_cuda_work_device(int noutput_items, const unsigned long* input_items,unsigned long* output_items) 
{
  // pointer for device memory
  float *d_idata = (float*)input_items[0];
  float *d_odata;
  unsigned int o_size = (noutput_items) * sizeof (float);

  hipMalloc( (void **) &d_odata, o_size);
  checkCUDAError("Malloc");
		
#define tpb 128
  int grid = noutput_items/tpb;
  if(noutput_items % tpb)
    grid++;
  dim3 dimGrid(grid);
  dim3 dimBlock(tpb);

  sub_const_ff_cuda_kernel<<<dimGrid, dimBlock>>>(d_idata, d_odata, .2, noutput_items);
  checkCUDAError("kernel execution 1");

  hipFree(d_idata);

  //now fill out the output output items array with the corresponding device pointers
  for(int i=0; i<noutput_items; i++){
    output_items[i]=(unsigned long)d_odata+i*8;
  }
}
