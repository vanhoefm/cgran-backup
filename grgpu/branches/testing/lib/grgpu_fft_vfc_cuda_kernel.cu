#include "hip/hip_runtime.h"
/* -*- cuda -*- */
/*
 * Copyright 2011 Free Software Foundation, Inc.
 * 
 * This file is part of GNU Radio
 * 
 * GNU Radio is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 3, or (at your option)
 * any later version.
 * 
 * GNU Radio is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with GNU Radio; see the file COPYING.  If not, write to
 * the Free Software Foundation, Inc., 51 Franklin Street,
 * Boston, MA 02110-1301, USA.
 *
 * This file was modified by William Plishker in 2011 for the GNU Radio 
 * support package GRGPU.  See www.cgran.org/wiki/GRGPU for more details. 
 */ 

#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <cutil_inline.h>
#include <grgpu_utils.h>
#include <hipfft/hipfft.h>


__global__ void fft_vfc_cuda_kernel(float* d_idata, float *d_odata, float k, int n)
{
  int idx = threadIdx.x+blockIdx.x*blockDim.x;
  if(idx<n)
    d_odata[idx]=d_idata[idx]+k;
}


void grgpu_fft_vfc_cuda_work_device(int noutput_items, const unsigned long* input_items,unsigned long* output_items, void **plan) 
{

  if(*plan==0x0){
    *plan = malloc(sizeof(hipfftHandle));
    hipfftPlan1d((hipfftHandle*)*plan, 1024, HIPFFT_R2C, 1);
  }


  // pointer for device memory
  float *d_idata = (float*)input_items[0];
  float *d_odata;
  //  unsigned int o_size = (noutput_items) * sizeof (float);

  // Use in-place buffer for performance
  d_odata = (float*)input_items[0];
  //  hipMalloc( (void **) &d_odata, o_size);
  //  checkCUDAError("Malloc");

  hipfftExecR2C(*(hipfftHandle*)*plan, (hipfftReal*)d_idata, (hipfftComplex*)d_odata);

  printf("%p\n", d_odata);
  //now fill out the output output items array with the corresponding device pointers
  for(int i=0; i<noutput_items; i+=1024){
    output_items[i]=(unsigned long)d_odata+i*8;
  }
}
